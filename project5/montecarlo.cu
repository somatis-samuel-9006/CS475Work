#include "hip/hip_runtime.h"
#include <stdio.h>
#define _USE_MATH_DEFINES
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include "hip/hip_runtime_api.h"
#include "helper_functions.h"
#include <hip/hip_runtime.h>


// setting the number of threads:
#ifndef NUMT
#define NUMT		    1
#endif

// setting the number of trials in the monte carlo simulation:
#ifndef NUMTRIALS
#define NUMTRIALS	2048
#endif


//block size in threads
#ifndef BLOCKSIZE
#define BLOCKSIZE 32
#endif

#define NUMBLOCKS  (NUMTRIALS / BLOCKSIZE)


// ranges for the random numbers:
const float GMIN =	20.0;	// ground distance in meters
const float GMAX =	30.0;	// ground distance in meters
const float HMIN =	10.0;	// cliff height in meters
const float HMAX =	40.0;	// cliff height in meters
const float DMIN  =	10.0;	// distance to castle in meters
const float DMAX  =	20.0;	// distance to castle in meters
const float VMIN  =	30.0;	// intial cnnonball velocity in meters / sec
const float VMAX  =	50.0;	// intial cnnonball velocity in meters / sec
const float THMIN = 	70.0;	// cannonball launch angle in degrees
const float THMAX =	80.0;	// cannonball launch angle in degrees

const float GRAVITY =	-9.8;	// acceleraion due to gravity in meters / sec^2
const float TOL = 5.0;		// tolerance in cannonball hitting the castle in meters
				// castle is destroyed if cannonball lands between d-TOL and d+TOL


float
Ranf( float low, float high )
{
        float r = (float) rand();               // 0 - RAND_MAX
        float t = r  /  (float) RAND_MAX;       // 0. - 1.

        return   low  +  t * ( high - low );
}

int
Ranf( int ilow, int ihigh )
{
        float low = (float)ilow;
        float high = ceil( (float)ihigh );

        return (int) Ranf(low,high);
}

// degrees-to-radians -- callable from the device:
__device__
float
Radians( float d )
{
        return (M_PI/180.f) * d;
}

void
TimeOfDaySeed( )
{
	struct tm y2k = { 0 };
	y2k.tm_hour = 0;   y2k.tm_min = 0; y2k.tm_sec = 0;
	y2k.tm_year = 100; y2k.tm_mon = 0; y2k.tm_mday = 1;

	time_t  timer;
	time( &timer );
	double seconds = difftime( timer, mktime(&y2k) );
	unsigned int seed = (unsigned int)( 1000.*seconds );    // milliseconds
	srand( seed );
}

void
CudaCheckError( )
{
        hipError_t e = hipGetLastError( );
        if( e != hipSuccess )
        {
                fprintf( stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e) );
        }
}
// the kernel:
__global__
void
MonteCarlo( float *dvs, float *dths, float *dgs, float *dhs, float *dds, int *dhits )
{
        unsigned int gid      = blockIdx.x*blockDim.x + threadIdx.x;

        // randomize everything:
        float v   = dvs[gid];
        float thr = Radians( dths[gid] );
        float vx  = v * cos(thr);
        float vy  = v * sin(thr);
        float  g  =  dgs[gid];
        float  h  =  dhs[gid];
        float  d  =  dds[gid];

        int numHits = 0;

        // see if the ball doesn't even reach the cliff:
        float t = -vy / ( 0.5*GRAVITY );
        float x = vx * t;
        //could put debug fprintf here if x <= g saying ball doesn't reach cliff
        if( x > g )
        {
            //see if ball hits vertical cliff face
            t = g / vx;
            float y = 0. + vy*t + 0.5*GRAVITY*(t*t);
            //could put fprintf ball hits cliff face here
            if( y > h)
            {
                //the ball hits the upper deck
                float a = 0.5*GRAVITY;
                float b = vy;
                float c = h * -1;
                float disc = b*b - 4.f*a*c;

                //if (disc < 0) exit(1)

                //successfully hits the ground above the cliff:
                disc = sqrtf(disc);
                float t1 = (-b + disc ) / ( 2.f*a );	// time to intersect high ground
				float t2 = (-b - disc ) / ( 2.f*a );	// time to intersect high ground

                //only care about the second intersection
                float tmax = t1;
                if( t2 > t1 )
                    tmax = t2;

                // how far does the ball land horizontlly from the edge of the cliff?
				float upperDist = vx * tmax  -  g;

                //see if ball hits the castle;
                if( fabs(upperDist - d) <= TOL)
                {
                    numHits = 1;
                }

            }
                           
        }

        dhits[gid] = numHits;
}


// these two #defines are just to label things
// other than that, they do nothing:
#define IN
#define OUT

int
main( int argc, char* argv[ ] )
{
        //double maxPerformance = 0.;
        TimeOfDaySeed( );

        int dev = findCudaDevice(argc, (const char **)argv);

        // better to define these here so that the rand() calls don't get into the thread timing:
        float *hvs   = new float [NUMTRIALS];
        float *hths  = new float [NUMTRIALS];
        float *hgs   = new float [NUMTRIALS];
        float *hhs   = new float [NUMTRIALS];
        float *hds   = new float [NUMTRIALS];
        int   *hhits = new int   [NUMTRIALS];

        // fill the random-value arrays:
        for(int n = 0; n < NUMTRIALS; n++)
        {
            hvs[n] = Ranf( VMIN,  VMAX );
            hths[n] = Ranf( THMIN, THMAX );
 		    hgs[n]  = Ranf(  GMIN,  GMAX );
 		    hhs[n]  = Ranf(  HMIN,  HMAX );
 		    hds[n]  = Ranf(  DMIN,  DMAX );
        }



        // allocate device memory:
        float *dvs, *dths, *dgs, *dhs, *dds;
        int   *dhits;

        hipMalloc( &dvs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dths,  NUMTRIALS*sizeof(float) );
        hipMalloc( &dgs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhs,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dds,   NUMTRIALS*sizeof(float) );
        hipMalloc( &dhits, NUMTRIALS*sizeof(int) );
        CudaCheckError( );

        // copy host memory to the device:
        hipMemcpy( dvs,  hvs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dths, hths, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dgs,  hgs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dhs,  hhs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        hipMemcpy( dds,  hds,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
        CudaCheckError( );

        // setup the execution parameters:
        dim3 grid( NUMBLOCKS, 1, 1 );
        dim3 threads( BLOCKSIZE, 1, 1 );

        // allocate cuda events that we'll use for timing:
        hipEvent_t start, stop;
        hipEventCreate( &start );
        hipEventCreate( &stop  );
        CudaCheckError( );

        // let the gpu go quiet:
        hipDeviceSynchronize( );

        // record the start event:
        hipEventRecord( start, NULL );
        CudaCheckError( );

        // execute the kernel:
        MonteCarlo<<< grid, threads >>>( IN dvs, IN dths, IN dgs, IN dhs, IN dds,   OUT dhits );

        // record the stop event:
        hipEventRecord( stop, NULL );
        CudaCheckError( );

        // wait for the stop event to complete:
        hipDeviceSynchronize( );
        hipEventSynchronize( stop );
        CudaCheckError( );

        float msecTotal = 0.0f;
        hipEventElapsedTime( &msecTotal, start, stop );
        CudaCheckError( );

        // compute and print the performance
        //change milliseconds to seconds
        float secTotal = msecTotal / 1000.;
        double megaTrialsPerSecond = (double)NUMTRIALS / secTotal / 1000000.;
        fprintf(stderr, "\n blocksize = %d    numtrials = %d    megatrials/sec = %6.2lf", BLOCKSIZE, NUMTRIALS, megaTrialsPerSecond);

	

        // copy result from the device to the host:
        hipMemcpy( hhits, dhits, NUMTRIALS*sizeof(int), hipMemcpyDeviceToHost );
        CudaCheckError( );

        // add up the hhits[ ] array: :

        int numberOfHits = 0;
        for(int j = 0; j < NUMTRIALS; j++)
        {
            numberOfHits = numberOfHits + hhits[j];
        }

        // compute and print the probability:
        float prob;
        prob = 100.f * (float)numberOfHits / (float)NUMTRIALS;
        fprintf(stderr, "    probability: = %6.2lf\n", prob);

	

        // clean up host memory:
        delete [ ] hvs;
        delete [ ] hths;
        delete [ ] hgs;
        delete [ ] hhs;
        delete [ ] hds;
        delete [ ] hhits;

        // clean up device memory:
        hipFree( dvs );
        hipFree( dths );
        hipFree( dgs );
        hipFree( dhs );
        hipFree( dds );
        hipFree( dhits );
        CudaCheckError( );

	return 0;
}

